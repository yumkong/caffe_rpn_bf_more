#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/rpn_center_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
//void Compute_distance_data_gpu(int nthreads, const int K, const Dtype* bottom,
	//      const Dtype* label,const Dtype* label_weight, const Dtype* center, Dtype* distance) 
//{
  //  for(int index = 0; index < nthreads; ++index) {
__global__ void Compute_distance_data_gpu(int nthreads, const int K, const Dtype* bottom,
	      const Dtype* label,const Dtype* label_weight, const Dtype* center, Dtype* distance) {
    CUDA_KERNEL_LOOP(index, nthreads) {
    int m = index / K;
    //int m = index;
    int k = index % K;  // no use here
    const int label_value = static_cast<int>(label[m]);
	//liu@0716 added
	const int label_weight_value = static_cast<int>(label_weight[m]);
    // distance(i) = x(i) - c_{y(i)}
    //liu@0716 changed
    //distance[index] = bottom[index] - center[label_value * K + k] ;
    // 0717 changed
    if(label_weight_value > 0)
    {
        distance[index] = (bottom[index] - center[label_value * K + k]);
    }
    // feat - center
    //caffe_gpu_sub(K, bottom + m * K, center + label_value * K, distance + m * K);
	// (feat - center) * label_weight (0 or 1)
	//caffe_gpu_scale(K, (Dtype)label_weight_value, distance + m * K, distance + m * K);
  }
}

template <typename Dtype>
__global__ void Compute_center_diff_gpu(int nthreads, const int M, const int K, 
        const Dtype* label,const Dtype* label_weight, const Dtype* distance, Dtype* variation_sum, 
        Dtype* center_diff) {
//void Compute_center_diff_gpu(int nthreads, const int M, const int K, 
//	  const Dtype* label,const Dtype* label_weight, const Dtype* distance, Dtype* variation_sum, 
//	  Dtype* center_diff) {
//  for(int index = 0; index < nthreads; ++index)
  CUDA_KERNEL_LOOP(index, nthreads) 
  {
    int count = 0;
    for (int m = 0; m < M; m++) {
      const int label_value = static_cast<int>(label[m]);
	  const int label_weight_value = static_cast<int>(label_weight[m]);
      if (label_weight_value > 0 && label_value == index) 
	  {
        ++count;
        for (int k = 0; k < K; k++) {
          variation_sum[index * K + k] -= distance[m * K + k];
        }
		//caffe_gpu_sub(K, variation_sum + index * K, distance + m * K, variation_sum + index * K);
      }
    }
    for (int k = 0; k < K; k++) {
      center_diff[index * K + k] = (Dtype)0.05 * variation_sum[index * K + k] /(count + (Dtype)1.);
    }
    //caffe_gpu_scale(K, (Dtype)1./(count + (Dtype)1.), variation_sum + index * K, center_diff + index * K);
  }
}


template <typename Dtype>
void RpnCenterLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int nthreads = M_ * K_;
  // liu 0719: initialized distance_ as 0 each time before using it
  caffe_gpu_set(M_ * K_, Dtype(0), distance_.mutable_gpu_data());
  //int nthreads = M_; // num x hei x wid
  //liu@0716 changed:  added bottom[2]->gpu_data()  (label_weight)
  Compute_distance_data_gpu<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
     CAFFE_CUDA_NUM_THREADS>>>(nthreads, K_, bottom[0]->gpu_data(), bottom[1]->gpu_data(),
      							bottom[2]->gpu_data(),
                                this->blobs_[0]->gpu_data(), distance_.mutable_gpu_data());
  Dtype dot;
  caffe_gpu_dot(M_ * K_, distance_.gpu_data(), distance_.gpu_data(), &dot);
  // ### liu: should change M_ to actual count
  Dtype valid_count;
  caffe_gpu_asum(M_, bottom[2]->gpu_data(), &valid_count);
  //Dtype loss = dot / M_ / Dtype(2);
  Dtype loss = dot / valid_count / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
  //LOG(INFO) << "dot = " << dot << ", valid_count = " << valid_count;
  //LOG(INFO) << "M_ = " << M_ << ", K_ = " << K_;
}

template <typename Dtype>
void RpnCenterLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  int nthreads = N_; // 2
  caffe_gpu_set(N_ * K_, Dtype(0), variation_sum_.mutable_gpu_data());
  //caffe_gpu_set(variation_sum_->count(), Dtype(0), variation_sum_.mutable_gpu_data());
  // liu: here do not need "label_weight" because distance is already 0 from forward_gpu computation
  // liu: NONONO: still need label weight to reduce computation
  Compute_center_diff_gpu<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, M_, K_, bottom[1]->gpu_data(),
                                bottom[2]->gpu_data(), distance_.gpu_data(), 
                                variation_sum_.mutable_gpu_data(), this->blobs_[0]->mutable_gpu_diff());
  // ### liu: should change M_ to actual count
  Dtype valid_count;
  //caffe_gpu_asum(M_, bottom[2]->gpu_data(), &valid_count);
  caffe_gpu_asum(bottom[2]->count(), bottom[2]->gpu_data(), &valid_count);
  if (propagate_down[0]) {
  	//liu@0716 changed
    //caffe_gpu_scale(M_ * K_, top[0]->cpu_diff()[0] / M_, 
    caffe_gpu_scale(M_ * K_, top[0]->cpu_diff()[0] / valid_count, 
                             distance_.gpu_data(), bottom[0]->mutable_gpu_diff());
  }
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(RpnCenterLossLayer);

}  // namespace caffe
